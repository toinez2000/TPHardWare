#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "functionMatrix.h"
#include "dense.h"




__global__ void Dense(float *input,float *weight,float *output,int SxI,int Sxo){
    
  int tabIP = blockIdx.x ;
  int ip = threadIdx.x;

  output[tabIP] += input[ip]*weight[tabIP*SxI+ip];

}


__global__ void DTanH(float *input,int SxI){
    
  
  int ip = threadIdx.x;

  input[ip] = tanh(input[ip]);

}

__global__ void softMax(float *input,int SxI,float *sum)
{
  int ip = threadIdx.x;

  input[ip] = input[ip]/sum[0];
}


__global__ void sumExpo(float *input,int SxI,float* sum)
{
  int ip = threadIdx.x;
  input[ip] = exp(input[ip]);
  sum[0] += input[ip];
}







//-----------------------------------------------------------------------------------

float* vectorGPUDense (float* input, float* Weight,float *output,int SxI,int Sxo,int ActiveFunction)
{

  float *sum;
  sum = 0;
	float *d_input, *d_Weight, *d_out, *d_sum;




	hipMalloc((void**)&d_input, sizeof(float)*SxI);
    	hipMalloc((void**)&d_Weight, sizeof(float)*SxI);
    	hipMalloc((void**)&d_out, sizeof(float)*Sxo);
      hipMalloc((void**)&d_sum, sizeof(float));
	
    	hipMemcpy(d_Weight, Weight, sizeof(float) *SxI, hipMemcpyHostToDevice);
    	hipMemcpy(d_input, input, sizeof(float) *SxI, hipMemcpyHostToDevice);
      hipMemcpy(d_out,output,sizeof(float) *Sxo,hipMemcpyHostToDevice);
      hipMemcpy(d_sum,sum,sizeof(float),hipMemcpyHostToDevice);

	// Main function

    dim3 blocks(Sxo); 
    dim3 threadsPerBlock(SxI);
 
      

    	 Dense<<<threadsPerBlock,blocks>>>(d_input, d_Weight, d_out, SxI,  Sxo);   //SIZE_C1_kernel
 
      if(ActiveFunction==0){DTanH<<<1,Sxo>>>(d_out,Sxo);} //TanH
      else{
          
          sumExpo<<<1,Sxo>>>(d_out,Sxo,d_sum);
          softMax<<<1,Sxo>>>(d_out,Sxo,d_sum);
      }

    	hipMemcpy(output, d_out, sizeof(float)*Sxo, hipMemcpyDeviceToHost);
	
      hipFree(d_sum);
    	hipFree(d_Weight);
    	hipFree(d_input);
    	hipFree(d_out);


}

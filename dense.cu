#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "functionMatrix.h"
#include "dense.h"

//fonction add bias 
__global__ void addBias(float *input,float *bias){
    
  int ip = threadIdx.x;

<<<<<<< HEAD
  output[ip] += bias[ip];
=======
  input[ip] += bias[ip];
>>>>>>> dvt

}



__global__ void Dense(float *input,float *weight,float *output,int SxI,int Sxo){
    
  int tabIP = blockIdx.x ; //for each output you add multiplication of weight and input

  for(int ip =0;ip < SxI;ip++){
    output[tabIP] += input[ip]*weight[tabIP*SxI+ip];}
>>>>>>> dvt

}

// active function TanH
__global__ void DTanH(float *input,int SxI){
    
  
  int ip = threadIdx.x;

  input[ip] = tanh(input[ip]);

}

//active function softMax
__global__ void softMax(float *input,int SxI,float sum)
{
  int ip = threadIdx.x;

  input[ip] = input[ip]/sum;
}

//function calcul the exp of each output
__global__ void Expo(float *input,int SxI)
{
  int ip = threadIdx.x;
  input[ip] = exp(input[ip]);
>>>>>>> dvt
}







//-----------------------------------------------------------------------------------

<<<<<<< HEAD
float* vectorGPUDense (float* input, float* Weight,float *output,int SxI,int Sxo,int ActiveFunction)
{

  float *sum;
  sum = 0;
	float *d_input, *d_Weight, *d_out, *d_sum;




	hipMalloc((void**)&d_input, sizeof(float)*SxI);
    	hipMalloc((void**)&d_Weight, sizeof(float)*SxI*Sxo+Sxo);
    	hipMalloc((void**)&d_out, sizeof(float)*Sxo);
      hipMalloc((void**)&d_sum, sizeof(float));
	
    	hipMemcpy(d_Weight, Weight, sizeof(float) *SxI*Sxo+Sxo, hipMemcpyHostToDevice);
    	hipMemcpy(d_input, input, sizeof(float) *SxI, hipMemcpyHostToDevice);
      hipMemcpy(d_out,output,sizeof(float) *Sxo,hipMemcpyHostToDevice);
      hipMemcpy(d_sum,sum,sizeof(float),hipMemcpyHostToDevice);
=======
float* vectorGPUDense (float* input, float* Weight,int SxI,int Sxo,int ActiveFunction)
{

  float sum, *output, *output0;

output= (float*)malloc(sizeof(float) *Sxo);
output0= (float*)malloc(sizeof(float) *Sxo);

  sum = 0;
	float *d_input, *d_Weight, *d_out;

    

//malloc GPU
	hipMalloc((void**)&d_input, sizeof(float)*SxI);
    	hipMalloc((void**)&d_Weight, sizeof(float)*(SxI*Sxo+Sxo));
    	hipMalloc((void**)&d_out, sizeof(float)*Sxo);
	
    	hipMemcpy(d_Weight, Weight, sizeof(float) *(SxI*Sxo+Sxo), hipMemcpyHostToDevice);
    	hipMemcpy(d_input, input, sizeof(float) *SxI, hipMemcpyHostToDevice);
      	hipMemcpy(d_out,output,sizeof(float) *Sxo,hipMemcpyHostToDevice);

	// Main function

    dim3 blocks(Sxo); 
<<<<<<< HEAD
    dim3 threadsPerBlock(SxI);
 
      

    	 Dense<<<threadsPerBlock,blocks>>>(d_input, d_Weight, d_out, SxI,  Sxo);   //SIZE_C1_kernel
 	 addBias<<<Sxo,1>>>(d_out,d_Weight+SxI*Sxo);
      if(ActiveFunction==0){DTanH<<<1,Sxo>>>(d_out,Sxo);} //TanH
      else{
          
          sumExpo<<<1,Sxo>>>(d_out,Sxo,d_sum);
          softMax<<<1,Sxo>>>(d_out,Sxo,d_sum);
=======
 
      

    	 Dense<<<blocks,1>>>(d_input, d_Weight, d_out, SxI,  Sxo);   //calcul each output of dense without bias
       hipDeviceSynchronize();
 	    addBias<<<Sxo,1>>>(d_out,d_Weight+SxI*Sxo); //add bias
      hipDeviceSynchronize();
      if(ActiveFunction==0){
      DTanH<<<1,Sxo>>>(d_out,Sxo); 
      hipDeviceSynchronize();
      
      } //TanH
      else{
          
          Expo<<<1,Sxo>>>(d_out,Sxo);
          hipDeviceSynchronize();
          hipMemcpy(output0, d_out, sizeof(float)*Sxo, hipMemcpyDeviceToHost);
          for(int j=0;j<Sxo;j++){
            sum =  sum+output0[j];
          }
          

          softMax<<<1,Sxo>>>(d_out,Sxo,sum);
          hipDeviceSynchronize();
>>>>>>> dvt
      }

    	hipMemcpy(output, d_out, sizeof(float)*Sxo, hipMemcpyDeviceToHost);
	//free
    	hipFree(d_Weight);
    	hipFree(d_input);
    	hipFree(d_out);

<<<<<<< HEAD

=======
  return output;
  
>>>>>>> dvt
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "functionMatrix.h"
#include "convolutionLayer.h"
#include "vector_add_N_P.h"

<<<<<<< HEAD
=======
__global__ void addLayerOutput(float* input, float* output, int SxI, int SzI, int SxK, int SzK) {
  // Calculer les indices de l'élément de sortie courant
  int xo = blockIdx.x * blockDim.x; // ligne de l'image de sortie pour une profondeur donné 
  int yo = threadIdx.x; // element de la ligne
  int zK = blockIdx.z ;
  int Sxo = (SxI-SxK+1);

	float value =0;
  
		// Appliquer le filtre sur chaque élément de l'image
	  for (int i = 0; i < SzI; i++) {
          	// Appliquer le filtre à l'élément de l'image courant
           value += input[xo+yo+  zK*Sxo*Sxo + i*SzK*Sxo*Sxo];
	      
	    }

  	  // Enregistrer la valeur de l'élément de sortie
    output[xo+ yo + zK*Sxo*Sxo] = value;
         
}




>>>>>>> dvt
__global__ void conv3D(float* input, float* kernel, float* output, int SxI, int SzI, int SxK, int SzK) {
  // Calculer les indices de l'élément de sortie courant
  int xo = blockIdx.x * blockDim.x; // ligne de l'image de sortie pour une profondeur donné 
  int yo = threadIdx.x; // element de la ligne
<<<<<<< HEAD
  int Sxo = (SxI-SxK+1)* (SxI-SxK+1);

  int zI = blockIdx.z ;   //profondeur du kernel à utiliser 
  int zK = threadIdx.z; // proffondeur de l'image input à utilisé 
=======
  int Sxo = (SxI-SxK+1);
  int Szo = SzK;

 

  int zK = blockIdx.z ;   //profondeur du kernel à utiliser 
  int zI = threadIdx.z; // proffondeur de l'image input à utilisé 
>>>>>>> dvt

  // Initialiser la valeur de l'élément de sortie à 0
  float value = 0.0;



  
		// Appliquer le filtre sur chaque élément de l'image
	  for (int i = 0; i < SxK; i++) {
	    for (int j = 0; j < SxK; j++) {
	        // Calculer les indices de l'élément de l'image à utiliser
			

			int decalage = blockIdx.x*(SxK-1);


<<<<<<< HEAD
          int xI = xo+ i*SxI + decalage +zI*(SxI*SxI);  // lien entre xI et x0
	        int yI = yo + j;
          	// Appliquer le filtre à l'élément de l'image courant
           value += input[xI+ yI] * kernel[i*SxK + j + zK*SxK*SxK+ zI*SzK*SxK*SxK];
          //printf("xo = %d,yo = %d, zI = %d, zk =%d , indiceInput = %d indiceKernel= %d \n",xo,yo,zI,zK,xI+ yI,i*SxK + j + zK*SxK*SxK);
=======
          int xI = xo+ i*SxI + decalage ;  // lien entre xI et x0
	        int yI = yo + j;
          	// Appliquer le filtre à l'élément de l'image courant
           value += input[xI+ yI+zI*(SxI*SxI)] * kernel[i*SxK + j + zK*SxK*SxK+ zI*SzK*SxK*SxK];
          //printf("xo = %d,yo = %d, zI = %d, zk =%d , indiceInput = %d indiceKernel= %d \n",xo,yo,zI,zK,xI+ yI,i*SxK + j + zK*SxK*SxK + zI*SzK*SxK*SxK);
>>>>>>> dvt
          //printf("xo = %f ,indiceK = %d \n",kernel[i*SxK + j + zK*SxK*SxK],i*SxK + j + zK*SxK*SxK);
	      }
	    }

  	  // Enregistrer la valeur de l'élément de sortie
<<<<<<< HEAD
    output[xo+ yo + (zK) *(SxI-SxK+1)*(SxI-SxK+1)] += value;
=======
    output[xo+ yo + zK*Sxo*Sxo + zI*Szo*Sxo*Sxo] = value;
	//output[7] = value;

	//if(xo ==0 && yo == 0 && zK ==1){printf("output = %f \n",output[zK*Sxo*Sxo]);}
	
>>>>>>> dvt
  
         
}


__global__ void addBias(float* M, float*bias, int SxI, int SzI) {
  // Calculer les indices de l'élément de sortie courant
  int deep = blockIdx.x; //  
  int yo = threadIdx.x; // 
 

<<<<<<< HEAD
     
    output[xo*SxI*SxI+ yo] += bias[deep];
=======
     //printf("bias = %f ", bias[deep]);
    M[deep*SxI*SxI+ yo] += bias[deep];
>>>>>>> dvt
  
         
}





float* vectorGPUConv1 (float* Kernel, float* input,int SxI,int SzI,int SxK,int SzK)
{

<<<<<<< HEAD
	float *out;
	float *d_input, *d_Kernel, *d_out;
  int Sxo = SxI-SxK+1;
  int Szo = SzK;

	out = (float*)malloc(sizeof(float) * Sxo*Sxo*Szo);

	hipMalloc((void**)&d_input, sizeof(float)*SxI*SxI*SzI);
    	hipMalloc((void**)&d_Kernel, sizeof(float)*SxK*SxK*SzK*SzI+SzK);
    	hipMalloc((void**)&d_out, sizeof(float)*Sxo*Sxo*Szo);

	
    	hipMemcpy(d_Kernel, Kernel, sizeof(float) *SxK*SxK*SzK*SzI+SzK, hipMemcpyHostToDevice);
=======
	float *out ;
	float *d_input, *d_Kernel, *d_out, *d_out0;
  int Sxo = SxI-SxK+1;
  int Szo = SzK;
    // printf("bias = %f ", Kernel[SzI*SxK*SxK*SzK+1]);

	out = (float*)malloc(sizeof(float) * Sxo*Sxo*Szo);
/*
	printf("C1 \n\n");
    print_matrix(out,Sxo*Szo*SzI,Sxo);

*/

	hipMalloc((void**)&d_input, sizeof(float)*SxI*SxI*SzI);
    	hipMalloc((void**)&d_Kernel, sizeof(float)*(SxK*SxK*SzK*SzI+SzK));
    	hipMalloc((void**)&d_out, sizeof(float)*Sxo*Sxo*Szo);

		hipMalloc((void**)&d_out0, sizeof(float)*Sxo*Sxo*Szo*SzI);

	
    	hipMemcpy(d_Kernel, Kernel, sizeof(float) *(SxK*SxK*SzK*SzI+SzK), hipMemcpyHostToDevice);
>>>>>>> dvt
    	hipMemcpy(d_input, input, sizeof(float) *SxI*SxI*SzI, hipMemcpyHostToDevice);

	// Main function
    	//int block_size = atoi(argv[2]);
    	//int grid_size = atoi(argv[1]);
 
<<<<<<< HEAD
    dim3 blocks( Sxo, 1, SzK ); 
    dim3 threadsPerBlock( Sxo, 1, SzI );
 
      

    	conv3D<<<threadsPerBlock,blocks>>>(d_input, d_Kernel, d_out, SxI, SzI, SxK,  SzK);   //SIZE_C1_kernel
	addBias<<<Sxo*Sxo,SzK>>>(d_out, d_Kernel+SxK*SxK*SzI*SzK, Sxo, SzK);
	
=======
    dim3 blocks( Sxo, 1, SzI ); 
    dim3 threadsPerBlock( Sxo, 1, SzK );
 
      

    	conv3D<<<threadsPerBlock,blocks>>>(d_input, d_Kernel, d_out0, SxI, SzI, SxK,  SzK);   //SIZE_C1_kernel
		hipDeviceSynchronize();
		dim3 blocks1( Sxo, 1, 1); 


		addLayerOutput<<<threadsPerBlock,blocks1>>>(d_out0, d_out,SxI, SzI,  SxK,  SzK);
		hipDeviceSynchronize();
		
		addBias<<<SzK,Sxo*Sxo>>>(d_out, d_Kernel+SxK*SxK*SzI*SzK, Sxo, SzK);
		hipDeviceSynchronize();
>>>>>>> dvt
    	hipMemcpy(out, d_out, sizeof(float)*Sxo*Sxo*Szo, hipMemcpyDeviceToHost);
	
	
    	hipFree(d_Kernel);
    	hipFree(d_input);
    	hipFree(d_out);
<<<<<<< HEAD
=======
		hipFree(d_out0);
>>>>>>> dvt

	return out;

}

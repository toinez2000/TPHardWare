#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "functionMatrix.h"
#include "convolutionLayer.h"
#include "vector_add_N_P.h"

<<<<<<< HEAD
=======
__global__ void addLayerOutput(float* input, float* output, int SxI, int SzI, int SxK, int SzK) {
  // Calculer les indices de l'élément de sortie courant
  int xo = blockIdx.x * blockDim.x; // ligne de l'image de sortie pour une Deep input et output donnée
  int yo = threadIdx.x; // element de la ligne pour une Deep input et output donnée
  int zK = blockIdx.z ; // Deep du kernel  
  int Sxo = (SxI-SxK+1); // Size output en x

	float value =0;
  
		// Appliquer le filtre sur chaque élément de l'image
	  for (int i = 0; i < SzI; i++) {
          	
           value += input[xo+yo+  zK*Sxo*Sxo + i*SzK*Sxo*Sxo];  // sum(Cx_data[x_c= const][y_c = const ][zK = const][zI]) 
	      
	    }

  	  // Enregistrer la valeur dans la sortie
    output[xo+ yo + zK*Sxo*Sxo] = value;
         
}




>>>>>>> dvt
__global__ void conv3D(float* input, float* kernel, float* output, int SxI, int SzI, int SxK, int SzK) {
  // Calculer les indices de l'élément de sortie courant
  int xo = blockIdx.x * blockDim.x; // ligne de l'image de sortie pour une profondeur donnée 
  int yo = threadIdx.x; // element de la ligne
  int Sxo = (SxI-SxK+1); //Size output x
  int Szo = SzK; //Deep output 

 

  int zK = blockIdx.z ;   //profondeur du kernel à utiliser 
  int zI = threadIdx.z; // proffondeur de l'image input à utiliser

  // Initialiser la valeur de l'élément de sortie à 0
  float value = 0.0;



  
		// Appliquer le filtre sur chaque élément de l'image
	  for (int i = 0; i < SxK; i++) {
	    for (int j = 0; j < SxK; j++) {
	        // Calculer les indices de l'élément de l'image à utiliser
			

			int decalage = blockIdx.x*(SxK-1);


<<<<<<< HEAD
          int xI = xo+ i*SxI + decalage +zI*(SxI*SxI);  // lien entre xI et x0
	        int yI = yo + j;
          	// Appliquer le filtre à l'élément de l'image courant
           value += input[xI+ yI] * kernel[i*SxK + j + zK*SxK*SxK+ zI*SzK*SxK*SxK];
          //printf("xo = %d,yo = %d, zI = %d, zk =%d , indiceInput = %d indiceKernel= %d \n",xo,yo,zI,zK,xI+ yI,i*SxK + j + zK*SxK*SxK);
=======
          int xI = xo+ i*SxI + decalage ;  // lien entre xI et x0
	        int yI = yo + j;
          	// Appliquer le filtre à l'élément de l'image courant
           value += input[xI+ yI+zI*(SxI*SxI)] * kernel[i*SxK + j + zK*SxK*SxK+ zI*SzK*SxK*SxK];
          //printf("xo = %d,yo = %d, zI = %d, zk =%d , indiceInput = %d indiceKernel= %d \n",xo,yo,zI,zK,xI+ yI,i*SxK + j + zK*SxK*SxK + zI*SzK*SxK*SxK);
>>>>>>> dvt
          //printf("xo = %f ,indiceK = %d \n",kernel[i*SxK + j + zK*SxK*SxK],i*SxK + j + zK*SxK*SxK);
	      }
	    }

  	  // Enregistrer la valeur
    output[xo+ yo + zK*Sxo*Sxo + zI*Szo*Sxo*Sxo] = value;     
}


__global__ void addBias(float* M, float*bias, int SxI, int SzI) {
  // Calculer les indices de l'élément de sortie courant
  int deep = blockIdx.x; //  profondeur
  int yo = threadIdx.x; // element 
 

<<<<<<< HEAD
     
    output[xo*SxI*SxI+ yo] += bias[deep];
=======
     //printf("bias = %f ", bias[deep]);
    M[deep*SxI*SxI+ yo] += bias[deep];
>>>>>>> dvt
  
         
}





float* vectorGPUConv1 (float* Kernel, float* input,int SxI,int SzI,int SxK,int SzK)
{

<<<<<<< HEAD
	float *out;
	float *d_input, *d_Kernel, *d_out;
  int Sxo = SxI-SxK+1;
  int Szo = SzK;

	out = (float*)malloc(sizeof(float) * Sxo*Sxo*Szo);

	hipMalloc((void**)&d_input, sizeof(float)*SxI*SxI*SzI);
    	hipMalloc((void**)&d_Kernel, sizeof(float)*SxK*SxK*SzK*SzI+SzK);
    	hipMalloc((void**)&d_out, sizeof(float)*Sxo*Sxo*Szo);

	
    	hipMemcpy(d_Kernel, Kernel, sizeof(float) *SxK*SxK*SzK*SzI+SzK, hipMemcpyHostToDevice);
=======
	float *out ;
	float *d_input, *d_Kernel, *d_out, *d_out0;  
  	int Sxo = SxI-SxK+1;  //size output x convolution layer
  	int Szo = SzK;


	out = (float*)malloc(sizeof(float) * Sxo*Sxo*Szo);
/*
	printf("C1 \n\n");
    print_matrix(out,Sxo*Szo*SzI,Sxo);
*/

	//réservation de mémoire CPU
	hipMalloc((void**)&d_input, sizeof(float)*SxI*SxI*SzI);  
    	hipMalloc((void**)&d_Kernel, sizeof(float)*(SxK*SxK*SzK*SzI+SzK));
    	hipMalloc((void**)&d_out, sizeof(float)*Sxo*Sxo*Szo);
	hipMalloc((void**)&d_out0, sizeof(float)*Sxo*Sxo*Szo*SzI);  //d_out0 output intermédiaire cette matrice contient l'output de la convolution Sxo*Sxo*Szo*SzI

	
    	hipMemcpy(d_Kernel, Kernel, sizeof(float) *(SxK*SxK*SzK*SzI+SzK), hipMemcpyHostToDevice);
>>>>>>> dvt
    	hipMemcpy(d_input, input, sizeof(float) *SxI*SxI*SzI, hipMemcpyHostToDevice);

	// Main function
    	//int block_size = atoi(argv[2]);
    	//int grid_size = atoi(argv[1]);
 
<<<<<<< HEAD
    dim3 blocks( Sxo, 1, SzK ); 
    dim3 threadsPerBlock( Sxo, 1, SzI );
 
      

    	conv3D<<<threadsPerBlock,blocks>>>(d_input, d_Kernel, d_out, SxI, SzI, SxK,  SzK);   //SIZE_C1_kernel
	addBias<<<Sxo*Sxo,SzK>>>(d_out, d_Kernel+SxK*SxK*SzI*SzK, Sxo, SzK);
	
=======
    dim3 blocks( Sxo, 1, SzI ); 
    dim3 threadsPerBlock( Sxo, 1, SzK );
 
      

    	conv3D<<<threadsPerBlock,blocks>>>(d_input, d_Kernel, d_out0, SxI, SzI, SxK,  SzK);  
		hipDeviceSynchronize();
		
		//printf("SzI = %d \n",SzI);
		if(SzI>1){

			dim3 blocks1( Sxo, 1, 1); 
			addLayerOutput<<<threadsPerBlock,blocks1>>>(d_out0, d_out,SxI, SzI,  SxK,  SzK);
			hipDeviceSynchronize();
		
			addBias<<<SzK,Sxo*Sxo>>>(d_out, d_Kernel+SxK*SxK*SzI*SzK, Sxo, SzK);
			hipDeviceSynchronize();
    		hipMemcpy(out, d_out, sizeof(float)*Sxo*Sxo*Szo, hipMemcpyDeviceToHost);
			}
		else{
			addBias<<<SzK,Sxo*Sxo>>>(d_out, d_Kernel+SxK*SxK*SzI*SzK, Sxo, SzK);
			hipDeviceSynchronize();
    		hipMemcpy(out, d_out0, sizeof(float)*Sxo*Sxo*Szo, hipMemcpyDeviceToHost);
		}
	
	//free
    	hipFree(d_Kernel);
    	hipFree(d_input);
    	hipFree(d_out);
	hipFree(d_out0);

	return out;

}

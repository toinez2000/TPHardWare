#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "functionMatrix.h"
#include "convolutionLayer.h"
#include "vector_add_N_P.h"

__global__ void addLayerOutput(float* input, float* output, int SxI, int SzI, int SxK, int SzK) {
  // Calculer les indices de l'élément de sortie courant
  int xo = blockIdx.x * blockDim.x; // ligne de l'image de sortie pour une profondeur donné 
  int yo = threadIdx.x; // element de la ligne
  int zK = blockIdx.z ;
  int Sxo = (SxI-SxK+1);

	float value =0;
  
		// Appliquer le filtre sur chaque élément de l'image
	  for (int i = 0; i < SzI; i++) {
          	// Appliquer le filtre à l'élément de l'image courant
           value += input[xo+yo+  zK*Sxo*Sxo + i*SzK*Sxo*Sxo];
	      
	    }

  	  // Enregistrer la valeur de l'élément de sortie
    output[xo+ yo + zK*Sxo*Sxo] = value;
         
}




__global__ void conv3D(float* input, float* kernel, float* output, int SxI, int SzI, int SxK, int SzK) {
  // Calculer les indices de l'élément de sortie courant
  int xo = blockIdx.x * blockDim.x; // ligne de l'image de sortie pour une profondeur donné 
  int yo = threadIdx.x; // element de la ligne
  int Sxo = (SxI-SxK+1);
  int Szo = SzK;

 

  int zK = blockIdx.z ;   //profondeur du kernel à utiliser 
  int zI = threadIdx.z; // proffondeur de l'image input à utilisé 

  // Initialiser la valeur de l'élément de sortie à 0
  float value = 0.0;



  
		// Appliquer le filtre sur chaque élément de l'image
	  for (int i = 0; i < SxK; i++) {
	    for (int j = 0; j < SxK; j++) {
	        // Calculer les indices de l'élément de l'image à utiliser
			

			int decalage = blockIdx.x*(SxK-1);


          int xI = xo+ i*SxI + decalage ;  // lien entre xI et x0
	        int yI = yo + j;
          	// Appliquer le filtre à l'élément de l'image courant
           value += input[xI+ yI+zI*(SxI*SxI)] * kernel[i*SxK + j + zK*SxK*SxK+ zI*SzK*SxK*SxK];
          //printf("xo = %d,yo = %d, zI = %d, zk =%d , indiceInput = %d indiceKernel= %d \n",xo,yo,zI,zK,xI+ yI,i*SxK + j + zK*SxK*SxK + zI*SzK*SxK*SxK);
          //printf("xo = %f ,indiceK = %d \n",kernel[i*SxK + j + zK*SxK*SxK],i*SxK + j + zK*SxK*SxK);
	      }
	    }

  	  // Enregistrer la valeur de l'élément de sortie
    output[xo+ yo + zK*Sxo*Sxo + zI*Szo*Sxo*Sxo] = value;
	//output[7] = value;

	//if(xo ==0 && yo == 0 && zK ==1){printf("output = %f \n",output[zK*Sxo*Sxo]);}
	
  
         
}


__global__ void addBias(float* M, float*bias, int SxI, int SzI) {
  // Calculer les indices de l'élément de sortie courant
  int deep = blockIdx.x; //  
  int yo = threadIdx.x; // 
 

     //printf("bias = %f ", bias[deep]);
    M[deep*SxI*SxI+ yo] += bias[deep];
  
         
}





float* vectorGPUConv1 (float* Kernel, float* input,int SxI,int SzI,int SxK,int SzK)
{

	float *out ;
	float *d_input, *d_Kernel, *d_out, *d_out0;
  int Sxo = SxI-SxK+1;
  int Szo = SzK;
    // printf("bias = %f ", Kernel[SzI*SxK*SxK*SzK+1]);

	out = (float*)malloc(sizeof(float) * Sxo*Sxo*Szo);
/*
	printf("C1 \n\n");
    print_matrix(out,Sxo*Szo*SzI,Sxo);

*/

	hipMalloc((void**)&d_input, sizeof(float)*SxI*SxI*SzI);
    	hipMalloc((void**)&d_Kernel, sizeof(float)*(SxK*SxK*SzK*SzI+SzK));
    	hipMalloc((void**)&d_out, sizeof(float)*Sxo*Sxo*Szo);

		hipMalloc((void**)&d_out0, sizeof(float)*Sxo*Sxo*Szo*SzI);

	
    	hipMemcpy(d_Kernel, Kernel, sizeof(float) *(SxK*SxK*SzK*SzI+SzK), hipMemcpyHostToDevice);
    	hipMemcpy(d_input, input, sizeof(float) *SxI*SxI*SzI, hipMemcpyHostToDevice);

	// Main function
    	//int block_size = atoi(argv[2]);
    	//int grid_size = atoi(argv[1]);
 
    dim3 blocks( Sxo, 1, SzI ); 
    dim3 threadsPerBlock( Sxo, 1, SzK );
 
      

    	conv3D<<<threadsPerBlock,blocks>>>(d_input, d_Kernel, d_out0, SxI, SzI, SxK,  SzK);   //SIZE_C1_kernel
		hipDeviceSynchronize();
		dim3 blocks1( Sxo, 1, 1); 


		addLayerOutput<<<threadsPerBlock,blocks1>>>(d_out0, d_out,SxI, SzI,  SxK,  SzK);
		hipDeviceSynchronize();
		
		addBias<<<SzK,Sxo*Sxo>>>(d_out, d_Kernel+SxK*SxK*SzI*SzK, Sxo, SzK);
		hipDeviceSynchronize();
    	hipMemcpy(out, d_out, sizeof(float)*Sxo*Sxo*Szo, hipMemcpyDeviceToHost);
	
	
    	hipFree(d_Kernel);
    	hipFree(d_input);
    	hipFree(d_out);
		hipFree(d_out0);

	return out;

}

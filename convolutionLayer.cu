#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "functionMatrix.h"
#include "convolutionLayer.h"

__global__ void conv3D(float* input, float* kernel, float* output, int SxI, int SzI, int SxK, int SzK) {
  // Calculer les indices de l'élément de sortie courant
  int xo = blockIdx.x * blockDim.x; // ligne de l'image de sortie pour une profondeur donné 
  int yo = threadIdx.x; // element de la ligne
  int Sxo = (SxI-SxK+1)* (SxI-SxK+1);

  int zI = blockIdx.z ;   //profondeur du kernel à utiliser 
  int zK = threadIdx.z; // proffondeur de l'image input à utilisé 

  // Initialiser la valeur de l'élément de sortie à 0
  float value = 0.0;



  
		// Appliquer le filtre sur chaque élément de l'image
	  for (int i = 0; i < SxK; i++) {
	    for (int j = 0; j < SxK; j++) {
	        // Calculer les indices de l'élément de l'image à utiliser
			

			int decalage = blockIdx.x*(SxK-1);


          int xI = xo+ i*SxI + decalage +zI*(SxI*SxI);  // lien entre xI et x0
	        int yI = yo + j;
          	// Appliquer le filtre à l'élément de l'image courant
           value += input[xI+ yI] * kernel[i*SxK + j + zK*SxK*SxK];
          //printf("xo = %d,yo = %d, zI = %d, zk =%d , indiceInput = %d indiceKernel= %d \n",xo,yo,zI,zK,xI+ yI,i*SxK + j + zK*SxK*SxK);
          //printf("xo = %f ,indiceK = %d \n",kernel[i*SxK + j + zK*SxK*SxK],i*SxK + j + zK*SxK*SxK);
	      }
	    }

  	  // Enregistrer la valeur de l'élément de sortie
    output[xo+ yo + (zK) *(SxI-SxK+1)*(SxI-SxK+1)] += value;
  
         
}



float* vectorGPUConv1 (float* Kernel, float* input,int SxI,int SzI,int SxK,int SzK)
{

	float *out;
	float *d_input, *d_Kernel, *d_out;
  int Sxo = SxI-SxK+1;
  int Szo = SzK*SzI;

	out = (float*)malloc(sizeof(float) * Sxo*Sxo*Szo);

	hipMalloc((void**)&d_input, sizeof(float)*SxI*SxI*SzI);
    	hipMalloc((void**)&d_Kernel, sizeof(float)*SxK*SxK*SzK);
    	hipMalloc((void**)&d_out, sizeof(float)*Sxo*Sxo*Szo);

	
    	hipMemcpy(d_Kernel, Kernel, sizeof(float) *SxK*SxK*SzK, hipMemcpyHostToDevice);
    	hipMemcpy(d_input, input, sizeof(float) *SxI*SxI*SzI, hipMemcpyHostToDevice);

	// Main function
    	//int block_size = atoi(argv[2]);
    	//int grid_size = atoi(argv[1]);
 
    dim3 blocks( Sxo, 1, SzK ); 
    dim3 threadsPerBlock( Sxo, 1, SzI );
 
      

    	conv3D<<<threadsPerBlock,blocks>>>(d_input, d_Kernel, d_out, SxI, SzI, SxK,  SzK);   //SIZE_C1_kernel

    	hipMemcpy(out, d_out, sizeof(float)*Sxo*Sxo*Szo, hipMemcpyDeviceToHost);
	
	
    	hipFree(d_Kernel);
    	hipFree(d_input);
    	hipFree(d_out);

	return out;

}

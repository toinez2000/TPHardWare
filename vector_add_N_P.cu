#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "vector_add_N_P.h"
#define M 2000
#define P 2000
#define N M*P
#define MAX_ERR 1e-6

__global__ void vectorAdd (float *out, float *a, float *b, int n, int p) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int Size = n*p;
    // Handling arbitrary vector size
    if (tid < Size){
        out[tid] = a[tid] + b[tid];
    }
}



float* vectorGPUAdd (float* a, float* b, int n, int p)
{

	float *out;
	float *d_a, *d_b, *d_out;

	out = (float*)malloc(sizeof(float) * n*p);

	hipMalloc((void**)&d_a, sizeof(float)*n*p);
    	hipMalloc((void**)&d_b, sizeof(float)*n*p);
    	hipMalloc((void**)&d_out, sizeof(float)*n*p);

	
    	hipMemcpy(d_a, a, sizeof(float) * n*p, hipMemcpyHostToDevice);
    	hipMemcpy(d_b, b, sizeof(float) * n*p, hipMemcpyHostToDevice);

	// Main function
    	//int block_size = atoi(argv[2]);
    	//int grid_size = atoi(argv[1]);
	int block_size = p;
    	int grid_size = n;
    	vectorAdd<<<grid_size,block_size>>>(d_out, d_a, d_b, n,p);

    	hipMemcpy(out, d_out, sizeof(float)*n*p, hipMemcpyDeviceToHost);
	
	
    	hipFree(d_a);
    	hipFree(d_b);
    	hipFree(d_out);

	return out;

}

/*
int main(int argc, char *argv[]){

    float *a, *b, *out;
 

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }


	//function add
	out = vectorGPUAdd(a, b, M, P);

    // Verification
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }

    //printf("Grid size : %d\n", grid_size);
    //printf("Block size : %d\n", block_size);
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n"); 


    free(a);
    free(b);
    free(out);
}
*/
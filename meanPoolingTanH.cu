#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "functionMatrix.h"
#include "meanPoolingTanH.h"






__device__ float activation_tanh(float M)
{
  
      M = tanh(M);
  
  
  return M; 
}

//----------------------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------------------




__global__ void meanPooling(float* input, float* output, int SxI, int SzI) {
  // Calculer les indices de l'élément de sortie courant
  int xo = blockIdx.x * blockDim.x; // ligne de l'image de sortie pour une profondeur donné 
  int yo = threadIdx.x; // element de la ligne 

  int zI = threadIdx.z ;   //profondeur du kernel à utiliser 

  // Initialiser la valeur de l'élément de sortie à 0
  float value = 0.0;

  //lien xo et xI

  
		// Appliquer le filtre sur chaque élément de l'image
	  for (int i = 0; i < 2; i++) {
	    for (int j = 0; j < 2; j++) {
	        // Calculer les indices de l'élément de l'image à utiliser
			


	        int xI = 2*xo+ i*SxI;  // lien entre xI et x0
	        int yI = 2*yo + j;
       

  		// Appliquer le filtre à l'élément de l'image courant
          value += input[xI+ yI + zI*SxI*SxI];
        }
    }



  // Enregistrer la valeur de l'élément de sortie

  
  output[xo+ yo + zI *SxI*SxI/4] = activation_tanh(value/4); // moyenne de quatre élément 
}



//-------------------------------------------------

float* vectorGPUMeanPooling(float*input,int SxI,int SzI){
    	float *out;
	    float *d_input, *d_out;

	    out = (float*)malloc(sizeof(float) *SxI*SxI*SzI/4);

	    hipMalloc((void**)&d_input, sizeof(float)*SxI*SxI*SzI);
      hipMalloc((void**)&d_out, sizeof(float)* SxI*SxI*SzI/4);
      hipMemcpy(d_input, input, sizeof(float) * SxI*SxI*SzI, hipMemcpyHostToDevice);

	// Main function
 
      dim3 blocks( SxI/2, 1, SzI ); 
      dim3 threadsPerBlock( SxI/2, 1, 1 );
 
    	meanPooling<<<threadsPerBlock,blocks>>>(d_input, d_out, SxI, SzI);

    	hipMemcpy(out, d_out, sizeof(float)*SxI*SxI*SzI/4 , hipMemcpyDeviceToHost);
	
	
    	hipFree(d_input);
    	hipFree(d_out);
	return out;

}

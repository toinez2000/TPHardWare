#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "functionMatrix.h"
#include "modele.h"
#include "readfileWeight.h"
#include "functionMatrix.h"
#include "dense.h"
#include "convolutionLayer.h"
#include "meanPoolingTanH.h"

#define SIZE_raw_data  32
#define DEEP_raw_data  32
#define SIZE_C1_kernel 5
#define DEEP_K1 6
#define SIZE_C1_data (SIZE_raw_data-SIZE_C1_kernel+1)
<<<<<<< HEAD
#define SIZE_S1_data (SIZE_C1_data/2)        //14
=======
#define SIZE_S1_data  (SIZE_C1_data/2)        //14
>>>>>>> dvt

#define SIZE_C2_kernel 5
#define DEEP_K2 16
#define SIZE_C2_data (SIZE_S1_data-SIZE_C2_kernel+1)
#define SIZE_S2_data (SIZE_C2_data/2)  



#define SIZE_S3_data 120
#define SIZE_S4_data 84
#define SIZE_S5_data 10




#define TANH 0
#define SOFTMAX 1







<<<<<<< HEAD
int modele(){

    float *raw_data, *C1_data,*S1_data,*C1_kernel;
=======
float* modele( float* raw_data){

    float *C1_data,*S1_data,*C1_kernel;
>>>>>>> dvt
    float *C2_data,*S2_data,*C2_kernel;
    float *S3_data,*WeightD1;
    float *S4_data,*WeightD2;
    float *S5_data,*WeightD3;

    // Allocate memory and mapping
    
    float*WeightVector = readfile();
    
    C1_kernel = WeightVector;
    C2_kernel = C1_kernel+SIZE_C1_kernel*SIZE_C1_kernel*DEEP_K1+DEEP_K1;
    WeightD1 = C2_kernel+SIZE_C2_kernel*SIZE_C2_kernel*DEEP_K2*DEEP_K1+DEEP_K2;
    WeightD2 = WeightD1+SIZE_S2_data*SIZE_S3_data+SIZE_S3_data;
    WeightD3 = WeightD2+SIZE_S3_data*SIZE_S4_data+SIZE_S4_data;
        
<<<<<<< HEAD
        
        
    /*
    raw_data   = init_matrix(SIZE_raw_data, SIZE_raw_data);
    C1_kernel   = init_matrix(SIZE_C1_kernel *DEEP_K1, SIZE_C1_kernel );


    
    for(int j=0; j<DEEP_K1;j++){
        
      C1_kernel[j*SIZE_C1_kernel*SIZE_C1_kernel]=1.0; 
      for(int i =1;i<SIZE_C1_kernel*SIZE_C1_kernel;i++)
      {
        C1_kernel[i+j*SIZE_C1_kernel*SIZE_C1_kernel]=0.0;
      }
    }
*/

/*
keras.layers.Conv2D(16, kernel_size=5, strides=1, activation='tanh', padding='valid'), #C3
    keras.layers.AveragePooling2D(), #S4
    keras.layers.Flatten(), #Flatten
    keras.layers.Dense(120, activation='tanh'), #C5
    keras.layers.Dense(84, activation='tanh'), #F6
    keras.layers.Dense(10, activation='softmax') #Output layer
    #define SIZE_S3_data 120
#define SIZE_S4_data 84
#define SIZE_S5_data 10
*/

=======

        
    //------------------------------------ test conv

/*

    raw_data   = init_matrix(SIZE_raw_data*DEEP_raw_data, SIZE_raw_data);

    
    C1_kernel   = init_matrix(SIZE_C1_kernel *DEEP_K1*DEEP_raw_data+DEEP_K1, SIZE_C1_kernel );


    for(int z=0; z<DEEP_raw_data;z++){
        for(int j=0; j<DEEP_K1;j++){
        
            C1_kernel[j*SIZE_C1_kernel*SIZE_C1_kernel+z*SIZE_C1_kernel*SIZE_C1_kernel*DEEP_K1]=z+1; 
            for(int i =1;i<SIZE_C1_kernel*SIZE_C1_kernel;i++){
                 C1_kernel[i+j*SIZE_C1_kernel*SIZE_C1_kernel+z*SIZE_C1_kernel*SIZE_C1_kernel*DEEP_K1]=0.0;
            }
        }
    }
    for(int b=0;b<DEEP_K1;b++)
    {
        C1_kernel[DEEP_raw_data*SIZE_C1_kernel*SIZE_C1_kernel*DEEP_K1+b]=b+1;
    }

*/

//---------------------TEST DENSE
 
 
 /*
 float* W;
 
 
 raw_data   = init_matrix(SIZE_raw_data, 1);

    
    W   = init_matrix(SIZE_S1_data*SIZE_raw_data+SIZE_S1_data,1 );


   
    for(int i=0; i<SIZE_S1_data;i++){
        
            W[i*SIZE_raw_data]=i; 
            W[i*SIZE_raw_data+1]=i; 
            for(int j =2;j<SIZE_raw_data;j++){
                 W[j+i*SIZE_raw_data]=0.0;
            }
        }
    
    for(int b=0;b<SIZE_S1_data;b++)
    {
        W[SIZE_S1_data*SIZE_raw_data+b]=b+1;
    }



S1_data =vectorGPUDense(raw_data,W,SIZE_raw_data,SIZE_S1_data,1);

    hipDeviceSynchronize();

*/


>>>>>>> dvt
    C1_data =vectorGPUConv1(C1_kernel, raw_data,SIZE_raw_data,DEEP_raw_data,SIZE_C1_kernel,DEEP_K1 );
    S1_data =vectorGPUMeanPooling(C1_data,SIZE_C1_data,DEEP_K1);


<<<<<<< HEAD
    /*C2_data =vectorGPUConv1(C2_kernel, S1_data,SIZE_S1_data,DEEP_K1,SIZE_C2_kernel,DEEP_K2 );
    S2_data =vectorGPUMeanPooling(C2_data,SIZE_C2_data,DEEP_K2*DEEP_K1);

    vectorGPUDense (S2_data,WeightD1,S3_data,SIZE_S2_data*SIZE_S2_data*DEEP_K2*DEEP_K1,SIZE_S3_data,TANH);
    vectorGPUDense (S3_data,WeightD2,S4_data,SIZE_S3_data,SIZE_S4_data,TANH);
    vectorGPUDense (S4_data,WeightD3,S5_data,SIZE_S4_data,SIZE_S5_data,SOFTMAX);

*/

=======
    C2_data =vectorGPUConv1(C2_kernel, S1_data,SIZE_S1_data,DEEP_K1,SIZE_C2_kernel,DEEP_K2 );
    S2_data =vectorGPUMeanPooling(C2_data,SIZE_C2_data,DEEP_K2*DEEP_K1);

  S3_data = vectorGPUDense (S2_data,WeightD1,SIZE_S2_data*SIZE_S2_data*DEEP_K2*DEEP_K1,SIZE_S3_data,TANH);
  S4_data = vectorGPUDense (S3_data,WeightD2,SIZE_S3_data,SIZE_S4_data,TANH);
  S5_data =vectorGPUDense (S4_data,WeightD3,SIZE_S4_data,SIZE_S5_data,SOFTMAX);


//vectorGPUDense(raw_data,C1_kernel,C1_data,SIZE_raw_data*SIZE_raw_data,C1_kernel*C1_kernel,TANH);
>>>>>>> dvt





<<<<<<< HEAD
    hipDeviceSynchronize();




    //veref
    printf("Kernel \n\n");
    print_matrix(C1_kernel,SIZE_C1_kernel*DEEP_K1,SIZE_C1_kernel);
    printf("raw_data \n\n");
    print_matrix(raw_data,SIZE_raw_data,SIZE_raw_data);
=======



    //veref CONV
    /*
    printf("Kernel \n\n");
    print_matrix(C1_kernel,SIZE_C1_kernel*DEEP_K1*DEEP_raw_data,SIZE_C1_kernel);
    printf("raw_data \n\n");
    print_matrix(raw_data,SIZE_raw_data*DEEP_raw_data,SIZE_raw_data);
>>>>>>> dvt
    printf("C1 \n\n");
    print_matrix(C1_data,SIZE_C1_data*DEEP_K1,SIZE_C1_data);
    printf("S1 \n\n");
     print_matrix(S1_data,SIZE_S1_data*DEEP_K1,SIZE_S1_data);
<<<<<<< HEAD

\\free
=======
*/
//------------------------------------------------- veref dense
/*
    printf("W \n\n");
    print_matrix(W,SIZE_raw_data*SIZE_S1_data+SIZE_S1_data,1);
    printf("raw_data \n\n");
    print_matrix(raw_data,SIZE_raw_data,1);
    printf("S1 \n\n");
     print_matrix(S1_data,SIZE_S1_data,1);
     */





//free
>>>>>>> dvt
    free(raw_data);
    free(WeightVector);
    free(S1_data);
    free(S2_data);
    free(S3_data);
    free(S4_data);
<<<<<<< HEAD
    free(S5_data);
  

    return 0;
=======
    //free(S5_data);

    return S5_data;
>>>>>>> dvt
}

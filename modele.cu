#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "functionMatrix.h"
#include "modele.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>


#include "functionMatrix.h"
#include "dense.h"
#include "convolutionLayer.h"
#include "meanPoolingTanH.h"

#define SIZE_raw_data  6  //32
#define DEEP_raw_data  1  //32
#define SIZE_C1_kernel 3   //5
#define DEEP_K1 2//6
#define SIZE_C1_data (SIZE_raw_data-SIZE_C1_kernel+1)
#define SIZE_S1_data (SIZE_C1_data/2)        //14

#define SIZE_C2_kernel 3   //5
#define DEEP_K2 6//16
#define SIZE_C2_data (SIZE_S1_data-SIZE_C2_kernel+1)
#define SIZE_S2_data (SIZE_C2_data/2)  



#define SIZE_S3_data 120
#define SIZE_S4_data 84
#define SIZE_S5_data 10




#define TANH 0
#define SOFTMAX 1







int modele(){

    float *raw_data, *C1_data,*S1_data,*C1_kernel;
    float *C2_data,*S2_data,*C2_kernel;
    float *S3_data,*WeightD1;
    float *S4_data,*WeightD2;
    float *S5_data,*WeightD3;

    // Allocate memory
    raw_data   = init_matrix(SIZE_raw_data, SIZE_raw_data);
    C1_kernel   = init_matrix(SIZE_C1_kernel *DEEP_K1, SIZE_C1_kernel );


    
    for(int j=0; j<DEEP_K1;j++){
        
      C1_kernel[j*SIZE_C1_kernel*SIZE_C1_kernel]=1.0; 
      for(int i =1;i<SIZE_C1_kernel*SIZE_C1_kernel;i++)
      {
        C1_kernel[i+j*SIZE_C1_kernel*SIZE_C1_kernel]=0.0;
      }
    }


/*
keras.layers.Conv2D(16, kernel_size=5, strides=1, activation='tanh', padding='valid'), #C3
    keras.layers.AveragePooling2D(), #S4
    keras.layers.Flatten(), #Flatten
    keras.layers.Dense(120, activation='tanh'), #C5
    keras.layers.Dense(84, activation='tanh'), #F6
    keras.layers.Dense(10, activation='softmax') #Output layer
    #define SIZE_S3_data 120
#define SIZE_S4_data 84
#define SIZE_S5_data 10
*/

    C1_data =vectorGPUConv1(C1_kernel, raw_data,SIZE_raw_data,DEEP_raw_data,SIZE_C1_kernel,DEEP_K1 );
    S1_data =vectorGPUMeanPooling(C1_data,SIZE_C1_data,DEEP_K1);


    /*C2_data =vectorGPUConv1(C2_kernel, S1_data,SIZE_S1_data,DEEP_K1,SIZE_C2_kernel,DEEP_K2 );
    S2_data =vectorGPUMeanPooling(C2_data,SIZE_C2_data,DEEP_K2*DEEP_K1);

    vectorGPUDense (S2_data,WeightD1,S3_data,SIZE_S2_data*SIZE_S2_data*DEEP_K2*DEEP_K1,SIZE_S3_data,TANH);
    vectorGPUDense (S3_data,WeightD2,S4_data,SIZE_S3_data,SIZE_S4_data,TANH);
    vectorGPUDense (S4_data,WeightD3,S5_data,SIZE_S4_data,SIZE_S5_data,SOFTMAX);

*/






    hipDeviceSynchronize();




    //veref
    printf("Kernel \n\n");
    print_matrix(C1_kernel,SIZE_C1_kernel*DEEP_K1,SIZE_C1_kernel);
    printf("raw_data \n\n");
    print_matrix(raw_data,SIZE_raw_data,SIZE_raw_data);
    printf("C1 \n\n");
    print_matrix(C1_data,SIZE_C1_data*DEEP_K1,SIZE_C1_data);
    printf("S1 \n\n");
     print_matrix(S1_data,SIZE_S1_data*DEEP_K1,SIZE_S1_data);


    free(raw_data);
    free(C1_data);
    free(S1_data);
    free(C1_kernel);

    return 0;
}

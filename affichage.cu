

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#define WIDTH 28
#define HEIGHT 28


void charBckgrndPrint(char *str, int rgb[3]){
  printf("\033[48;2;%d;%d;%dm", rgb[0], rgb[1], rgb[2]);
  printf("%s\033[0m",str);
}

void imgColorPrint(int height, int width, int ***img){
  int row, col;
  char *str="  ";
  for(row=0; row<height; row++){
    for(col=0; col<width; col++){
      charBckgrndPrint(str,img[row][col]);
    }
    printf("\n");
  }
}

int mainAffiche() {
  int i, j;
  int ***img;
  int color[3]={255,0,0};
  unsigned int magic, nbImg, nbRows, nbCols;
  unsigned char val;
  FILE *fptr;

  // Malloc image
  img = (int ***)malloc(HEIGHT*sizeof(int **));
  for(i=0; i<HEIGHT; i++){
    img[i]= (int **)malloc(WIDTH*sizeof(int *));
    for(j=0; j<WIDTH; j++){
      img[i][j] = (int *)malloc(sizeof(int)*3);
    }
  }

  //Open File
  if((fptr = fopen("train-images.idx3-ubyte","rb")) == NULL){
    printf("Can't open file");
    exit(1);
  }

  //Read File
  fread(&magic, sizeof(int), 1, fptr);
  fread(&nbImg, sizeof(int), 1, fptr);
  fread(&nbRows, sizeof(int), 1, fptr);
  fread(&nbCols, sizeof(int), 1, fptr);
/*
  printf("Nb Magic : %u \n", magic);
  printf("Nb Img : %u \n", nbImg);
  printf("Nb Rows : %u \n", nbRows);
  printf("Nb Cols : %u \n", nbCols);
*/
  for(i=0; i<HEIGHT; i++){
    for(j=0; j<WIDTH; j++){ 
      fread(&val, sizeof(unsigned char), 1, fptr);  
      img[i][j][0]=(int)val*color[0]/255;
      img[i][j][1]=(int)val*color[1]/255;
      img[i][j][2]=(int)val*color[2]/255;
    }
  }

  imgColorPrint(HEIGHT, WIDTH, img);

  // setup image grayscale
  for(i=0; i<HEIGHT; i++){
    for(j=0; j<WIDTH; j++){
        img[i][j][0] = ((i+j)*4)%255;
        img[i][j][1] = ((i+j)*4)%255;
        img[i][j][2] = ((i+j)*4)%255;
    }
  }

  // print image
  imgColorPrint(HEIGHT, WIDTH, img);
  return 0;
  //exit(EXIT_SUCCESS);
}



float* readImage() {

  printf("read input mnist \n");
  int i, j;
  float* output;
  int ***img;
  int color[3]={255,0,0};
  unsigned int magic, nbImg, nbRows, nbCols;
  unsigned char val;
  FILE *fptr;


 

  //Open File
  if((fptr = fopen("train-images.idx3-ubyte","rb")) == NULL){
    printf("Can't open file");
    exit(1);
  }

  //Read File
  fread(&magic, sizeof(int), 1, fptr);
  fread(&nbImg, sizeof(int), 1, fptr);
  fread(&nbRows, sizeof(int), 1, fptr);
  fread(&nbCols, sizeof(int), 1, fptr);
/*
  printf("Nb Magic : %u \n", magic);
  printf("Nb Img : %u \n", nbImg);
  printf("Nb Rows : %u \n", nbRows);
  printf("Nb Cols : %u \n", nbCols);
*/

 output = (float *)malloc(sizeof(float)*32*32);
  float max =0;

  //flatten img and reshape 32*32
 
  for(int x=0;x<28;x++)
        {
            for(int y=0; y<28;y++){
                fread(&val, sizeof(unsigned char), 1, fptr);
                output[x+y*28] = (int)val*color[0]/255;

                if (max<output[x+y*28])
                { max = output[x+y*28];}
            }
        }
        for(int z=0;z<240;z++){
            output[28*28+z] = 0;

        }

/*
        for(int x=0;x<32*32;x++){
          output[x]= ((output[x]/max)-0.5)*2; //normalize
        }
*/
  
  return output;
}

#include "hip/hip_runtime.h"

# include <stdio.h>
//#include "functionMatrix.h"



#include <stdlib.h>
//#include <stdio.h>
//#include "functionMatrix.h"



__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}



void c_hello(){
    printf("Hello World!\n");
}







/* int main() {
    c_hello();
    return 0;
}
*/








float* init_matrix(int sizeX, int sizeY)
{
    float* matrice  = (float*) malloc(sizeX*sizeY*sizeof(float));
    time_t t;
    srand((unsigned) time(&t));

    for(int x=0;x<sizeX;x++){
        for(int y =0; y<sizeY; y++){
            int u = rand();
            matrice[x*sizeY+y]=((float(u)/RAND_MAX)-0.5);
            
        }
    }
    return matrice;
}


void print_matrix(float* matrice,int sizeX,int sizeY)
{

    for(int x=0;x<sizeX;x++){
        for(int y =0; y<sizeY; y++){
           
                printf(" %f ",matrice[x*sizeY+y]) ;
        }
        printf("\n");
    }
}






float* addMatrix(float* matriceA , float* matriceB, int sizeX, int sizeY)
{
     float* matrice  = (float*) malloc(sizeX*sizeY*sizeof(float));
     for(int x=0;x<sizeX;x++){
        for(int y =0; y<sizeY; y++){
           matrice[x*sizeY+y] = matriceA[x*sizeY+y] + matriceB[x*sizeY+y];
                
        }
    }
    return matrice;
}





float* multiMatrix(float* matriceA , float* matriceB, int sizeXa, int sizeYa,int sizeXb,int sizeYb)
{
    //assert(sizeXa==sizeYb);
     float* matrice  = (float*) malloc(sizeXb*sizeYa*sizeof(float));
     float u;
     for(int x=0;x<sizeXb;x++){
        for(int y =0; y<sizeYa; y++){
            u=0;
            for(int j=0;j<sizeXa;j++){
                u+=matriceA[j*sizeYa+y]*matriceB[x*sizeYb+j];
            }
           matrice[x*sizeYa+y] =u;
                
        }
    }
    return matrice;


}





int main() {
 
    c_hello();
    cuda_hello<<<1,1>>>(); 

    float* matriceA =  init_matrix(2,2);
    matriceA[0]=1;
    matriceA[1]=1;
    matriceA[2]=1;
    matriceA[3]=1;
    float* matriceB =  init_matrix(2,2);

    //float* matriceC =  multiMatrix(matriceA ,matriceA, 2,2,2,2);

    float* matriceC =  addMatrix(matriceA ,matriceA, 2,2);
    
    print_matrix(matriceC,2,2);




    hipDeviceSynchronize();
    return 0;
}


#include "hip/hip_runtime.h"

# include <stdio.h>

#include "vector_add_N_P.h"
#include "vector_multi_N_P.h"

#include <stdlib.h>

#include "functionMatrix.h"

#include "modele.h"
#include "readfileWeight.h"
#include "affichage.h"

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}



void c_hello(){
    printf("Hello World!\n");
}







/* int main() {
    c_hello();
    return 0;
}
*/


int main() {
 

//readfile();


 /*
    c_hello();
    cuda_hello<<<1,1>>>(); 

    float* matriceA =  init_matrix(1000,1000);
    matriceA[0]=1;
    matriceA[1]=2;
    matriceA[2]=1;
    matriceA[3]=1;
    float* matriceB =  init_matrix(2,2);

    float* matriceC =  multiMatrix(matriceA ,matriceA, 1000,1000,1000,1000);

    //float* matriceC =  addMatrix(matriceA ,matriceA, 2,2);


//float* matriceC = vectorGPUAdd (matriceA, matriceA, 2, 2);
//float* matriceC = multimatriceGPU(matriceA, matriceA, 1000);

    
    
    

*/

    mainAffiche();
    float* matriceinput =  readImage();


printf("BeginMain \n");

/*
    print_matrix(matriceinput,32,32);*/

    float*output =  modele(matriceinput);
    //hipDeviceSynchronize();
    //print_matrix(matriceC,10,10);

    printf("output \n");
    print_matrix(output,10,1);




    return 0;
}

// Pour multiplication GPUsur N=1000 time executiuon = 0.236s
// Pour multiplication CPU sur N=1000 time executiuon = 8.650s

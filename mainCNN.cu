#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "functionMatrix.h"


#define SIZE_raw_data 32
#define DEEP 6
#define SIZE_C1_data 28
#define SIZE_S1_data 14
#define SIZE_C1_kernel 5






__global__ void convolutionLayer(float* C1_data,float* C1_kernel, float* raw_data){

    int tind = blockIdx.x*blockDim.x+threadIdx.x;
    // tid = ligne*tailleLigne+elementLigne


    for(int j=0;j< SIZE_C1_kernel;j++){
    for(int i=0; i < SIZE_C1_kernel; i++){


        
            C1_data[tind]+= raw_data[tind+i*SIZE_C1_kernel+j-(tind%(SIZE_C1_data*SIZE_C1_data))*(SIZE_C1_data*SIZE_C1_data)]*C1_kernel[(i*SIZE_C1_kernel+j)*(tind%(SIZE_C1_data*SIZE_C1_data))*(SIZE_C1_kernel*SIZE_C1_kernel)];



        }
    }

   


}

float* vectorGPUConv1 (float* Kernel, float* input)
{

	float *out;
	float *d_input, *d_Kernel, *d_out;

	out = (float*)malloc(sizeof(float) * SIZE_C1_data*SIZE_C1_data);

	hipMalloc((void**)&d_input, sizeof(float)*SIZE_raw_data*SIZE_raw_data);
    	hipMalloc((void**)&d_Kernel, sizeof(float)*SIZE_C1_kernel*SIZE_C1_kernel);
    	hipMalloc((void**)&d_out, sizeof(float)*SIZE_C1_data *SIZE_C1_data );

	
    	hipMemcpy(d_Kernel, Kernel, sizeof(float) * SIZE_C1_kernel*SIZE_C1_kernel, hipMemcpyHostToDevice);
    	hipMemcpy(d_input, input, sizeof(float) * SIZE_raw_data*SIZE_raw_data, hipMemcpyHostToDevice);

	// Main function
    	//int block_size = atoi(argv[2]);
    	//int grid_size = atoi(argv[1]);
	int block_size = DEEP*SIZE_C1_data ;
    	int grid_size = SIZE_C1_data ;
    	convolutionLayer<<<grid_size,block_size>>>(d_out,d_Kernel, d_input);

    	hipMemcpy(out, d_out, sizeof(float)*SIZE_C1_data *SIZE_C1_data , hipMemcpyDeviceToHost);
	
	
    	hipFree(d_Kernel);
    	hipFree(d_input);
    	hipFree(d_out);

	return out;

}


int main2(){

    float *raw_data, *C1_data,*S1_data,*C1_kernel;


    // Allocate memory
    raw_data   = init_matrix(SIZE_raw_data, SIZE_raw_data);
    C1_data   = (float*)malloc(sizeof(float) * DEEP*SIZE_C1_data*SIZE_C1_data);
    S1_data   = (float*)malloc(sizeof(float) * DEEP*SIZE_S1_data*SIZE_S1_data);
    C1_kernel   = init_matrix(SIZE_C1_kernel *DEEP, SIZE_C1_kernel );


    




//Une matrice float raw_data de taille 32x32 initialisé avec des valeurs comprises entre 0 et 1, correspondant à nos données d'entrée.
//Une matrice float C1_data de taille 6x28x28 initialisé à 0 qui prendra les valeurs de sortie de la convolution 2D. C1 correspond aux données après la première Convolution.
//Une matrice float S1_data de taille 6x14x14 intialisé à 0 qui prendra les valeurs de sortie du sous-échantillonnage. S1 correspond aux données après le premier Sous-échantillonnage.
//Une matrice float C1_kernel de taille 6x5x5 initialisé à des valeurs comprises entre 0 et 1 correspondant à nos premiers noyaux de convolution.





    float* matriceC =vectorGPUConv1(C1_kernel, raw_data );
    hipDeviceSynchronize();
    print_matrix(raw_data,28,28);


    free(raw_data);
    free(C1_data);
    free(S1_data);
    free(C1_kernel);
    free(matriceC);

    return 0;
}




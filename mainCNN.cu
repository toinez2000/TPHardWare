#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "functionMatrix.h"


#define SIZE_raw_data  4  //32
#define SIZE_C1_kernel 2   //5
#define DEEP 2//6
#define SIZE_C1_data (SIZE_raw_data-SIZE_C1_kernel+1)
#define SIZE_S1_data 14







__global__ void convolutionLayer(float* C1_data,float* C1_kernel, float* raw_data){

    int tind = blockIdx.x*blockDim.x+threadIdx.x;
    // tid = ligne*tailleLigne+elementLigne
    
    int deepN = ((int)tind)/((int)(SIZE_C1_data*SIZE_C1_data));
    //printf("deepN = %d,tind = %d, SIZE_C1_data = %d \n",deepN,tind,SIZE_C1_data*SIZE_C1_data);

    int a = tind+(SIZE_C1_kernel-1)*(blockIdx.x);


    for(int i=0;i< SIZE_C1_kernel;i++){
    for(int j=0; j < SIZE_C1_kernel; j++){


            
            
           
          

            
            C1_data[tind]+= raw_data[a+i*SIZE_raw_data+j-deepN*(SIZE_raw_data*SIZE_raw_data)]*C1_kernel[(i*SIZE_C1_kernel+j)+deepN*(SIZE_C1_kernel*SIZE_C1_kernel)];
            //printf("%f \n",C1_data[tind]);
            
            
            if(tind == 5 && i == 0 && j ==0){
            printf("a = %d \n",a);
            printf("deepN = %d \n",deepN);
            printf("SIZE_C1_data  = %d \n",SIZE_C1_data );
            printf("%f \n",raw_data[a+i*SIZE_raw_data+j-deepN*(SIZE_raw_data*SIZE_raw_data)]);
            }


        }
    }
    //printf("%d \n",tind);

   


}

float* vectorGPUConv1 (float* Kernel, float* input)
{

	float *out;
	float *d_input, *d_Kernel, *d_out;

	out = (float*)malloc(sizeof(float) * SIZE_C1_data*SIZE_C1_data*DEEP);

	hipMalloc((void**)&d_input, sizeof(float)*SIZE_raw_data*SIZE_raw_data);
    	hipMalloc((void**)&d_Kernel, sizeof(float)*SIZE_C1_kernel*SIZE_C1_kernel*DEEP);
    	hipMalloc((void**)&d_out, sizeof(float)*SIZE_C1_data *SIZE_C1_data*DEEP);

	
    	hipMemcpy(d_Kernel, Kernel, sizeof(float) * SIZE_C1_kernel*SIZE_C1_kernel*DEEP, hipMemcpyHostToDevice);
    	hipMemcpy(d_input, input, sizeof(float) * SIZE_raw_data*SIZE_raw_data, hipMemcpyHostToDevice);

	// Main function
    	//int block_size = atoi(argv[2]);
    	//int grid_size = atoi(argv[1]);
	int block_size = DEEP*SIZE_C1_data ;
    	int grid_size =SIZE_C1_data ;
    	convolutionLayer<<<grid_size,block_size>>>(d_out,d_Kernel, d_input);

    	hipMemcpy(out, d_out, sizeof(float)*SIZE_C1_data *SIZE_C1_data*DEEP , hipMemcpyDeviceToHost);
	
	
    	hipFree(d_Kernel);
    	hipFree(d_input);
    	hipFree(d_out);

	return out;

}


int main2(){

    float *raw_data, *C1_data,*S1_data,*C1_kernel;


    // Allocate memory
    raw_data   = init_matrix(SIZE_raw_data, SIZE_raw_data);
    C1_data   = (float*)malloc(sizeof(float) * DEEP*SIZE_C1_data*SIZE_C1_data);
    S1_data   = (float*)malloc(sizeof(float) * DEEP*SIZE_S1_data*SIZE_S1_data);
    C1_kernel   = init_matrix(SIZE_C1_kernel *DEEP, SIZE_C1_kernel );
    C1_kernel[0]=1; 
    C1_kernel[1]=0;
    C1_kernel[2]=0;
    C1_kernel[3]=0;

    C1_kernel[4]=1; 
    C1_kernel[5]=0;
    C1_kernel[6]=0;
    C1_kernel[7]=0;



    




//Une matrice float raw_data de taille 32x32 initialisé avec des valeurs comprises entre 0 et 1, correspondant à nos données d'entrée.
//Une matrice float C1_data de taille 6x28x28 initialisé à 0 qui prendra les valeurs de sortie de la convolution 2D. C1 correspond aux données après la première Convolution.
//Une matrice float S1_data de taille 6x14x14 intialisé à 0 qui prendra les valeurs de sortie du sous-échantillonnage. S1 correspond aux données après le premier Sous-échantillonnage.
//Une matrice float C1_kernel de taille 6x5x5 initialisé à des valeurs comprises entre 0 et 1 correspondant à nos premiers noyaux de convolution.





    float* matriceC =vectorGPUConv1(C1_kernel, raw_data );
    hipDeviceSynchronize();
    print_matrix(raw_data,4,4);
    printf("\n");
    print_matrix(matriceC,SIZE_C1_data*DEEP,SIZE_C1_data);


    free(raw_data);
    free(C1_data);
    free(S1_data);
    free(C1_kernel);
    free(matriceC);

    return 0;
}




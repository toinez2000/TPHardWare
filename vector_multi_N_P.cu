#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "vector_multi_N_P.h"

#define M 2000
#define P 2000
#define N M*P
#define MAX_ERR 1e-6

__global__ void MatriceNxN_multi(float *out, float *a, float *b, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int Size = n*n;

// Handling arbitrary vector size
 	if (tid < Size){
		for(int i =0; i < n; i++){
			out[tid] = out[tid]+ a[blockIdx.x*blockDim.x+i] * b[i*blockDim.x+threadIdx.x];

		}
    	}
}


__host__ float* multimatriceGPU(float* a,float* b, int n){

	float *out;
	float *d_a, *d_b, *d_out;

	out = (float*)malloc(sizeof(float) * n*n);

	hipMalloc((void**)&d_a, sizeof(float)*n*n);
    	hipMalloc((void**)&d_b, sizeof(float)*n*n);
    	hipMalloc((void**)&d_out, sizeof(float)*n*n);

	
    	hipMemcpy(d_a, a, sizeof(float) * n*n, hipMemcpyHostToDevice);
    	hipMemcpy(d_b, b, sizeof(float) * n*n, hipMemcpyHostToDevice);

	// Main function
    	//int block_size = atoi(argv[2]);
    	//int grid_size = atoi(argv[1]);
		int block_size = n;
    	int grid_size = n;
    	MatriceNxN_multi<<<grid_size,block_size>>>(d_out, d_a, d_b, n);

    	hipMemcpy(out, d_out, sizeof(float)*n*n, hipMemcpyDeviceToHost);
	
	
    	hipFree(d_a);
    	hipFree(d_b);
    	hipFree(d_out);

	return out;

}




/*
int main1(int argc, char *argv[]){

    float *a, *b;


    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);


    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }


float* out =  multimatriceGPU(a,b, N);


    // Verification
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }

   
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");


    free(a);
    free(b);
    free(out);

	return 0;
}
*/